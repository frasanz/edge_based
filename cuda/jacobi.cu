/* This is the jacobi relaxation method on GPU, edge based */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

typedef struct {
  double function_u[3];
  double function_v[3];
  double function_f[3];
} triangle;



int main(){
  triangle ** h_grid;
  triangle ** d_grid;
  int size=5;
  int i,j;

  int triangles_base=(int)pow(2,size);

  /* Host allocation */
  h_grid = (triangle **)malloc(triangles_base*sizeof(triangle*));
  for(i=0;i<triangles_base;i++){
    h_grid[i]=(triangle*)malloc((triangles_base-i)*sizeof(triangle));
  }


  /* Host free */
  for(i=0;i<triangles_base;j++){
    free(h_grid[i]);
  }
  free(h_grid);
  return 0;
}
