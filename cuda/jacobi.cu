/* This is the jacobi relaxation method on GPU, edge based */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

typedef struct {
  double function_u[3];
  double function_u_new[3];
  double function_v[3];
  double function_f[3];
} triangle;

void initialize_grid(triangle ** h_grid, int size){
  int i,j,k;
  int triangles_base=(int)pow(2,size);
  for(i=0;i<triangles_base;i++){
    for(j=0;j<triangles_base-i;j++){
      for(k=0;k<3;k++){
        h_grid[i][j].function_u[k]=0.0;
        h_grid[i][j].function_u_new[k]=0.0;
        h_grid[i][j].function_v[k]=0.0;
        h_grid[i][j].function_f[k]=0.0;
      }
    }
  }
  for(i=1;i<triangles_base-1;i++){
    for(j=1;j<triangles_base-i-1;j++){
      for(k=0;k<3;k++){
        h_grid[i][j].function_u[k]=10*sin(i+10*j+20*k);
      }
    }
  }
}

void smoother(triangle ** h_grid, int size){
  int i,j,k;
}

int main(){
  triangle ** h_grid;
  triangle ** d_grid;
  int size=5;
  int i;
  int triangles_base=(int)pow(2,size);

  /* Host allocation */
  h_grid = (triangle **)malloc(triangles_base*sizeof(triangle*));
  for(i=0;i<triangles_base;i++){
    h_grid[i]=(triangle*)malloc((triangles_base-i)*sizeof(triangle));
  }

  /* Initialization */
  initialize_grid(h_grid,size);


  /* Host free */
  for(i=0;i<triangles_base;i++){
    free(h_grid[i]);
  }

  free(h_grid);
  return 0;
}
